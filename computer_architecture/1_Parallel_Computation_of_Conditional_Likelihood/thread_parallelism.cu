
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define N 1000000
#define T 256

/*
<<<B, T>>>

gridDim.x = B
blockDim.x = T
blockIdx.x = 0 ... B - 1
threadIdx.x = 0 ... T - 1
*/


/*
clP - Cond0tional Likelihood of Parents (1x6)
clC - Conditional Likelihood of Children (1x12)
clPC - Transition Probability of Parent -> Child (1x12) 
*/

__global__ void compute_parent_likelihood (float *clP, float *clC, float *tiPC) {
    for (int p_id = threadIdx.x; p_id < 6 * N; p_id += T) {
        int l = p_id * 2, r = p_id * 2 + 1;

        clP[p_id] = tiPC[l] * clC[l] + tiPC[r] * clC[r];
    }
}

int main() {
    // Define and allocate host memory
    float *clP, *clC, *tiPC;
    
    clP = (float*) malloc(sizeof(float) * 6 * N);
    clC = (float*) malloc(sizeof(float) * 12 * N);
    tiPC = (float*) malloc(sizeof(float) * 12 * N);

    // Define and allocate device memory
    float *d_clP, *d_clC, *d_tiPC;

    hipMalloc((void**) &d_clP, sizeof(float) * 6 * N);
    hipMalloc((void**) &d_clC, sizeof(float) * 12 * N);
    hipMalloc((void**) &d_tiPC, sizeof(float) * 12 * N);

    // Initialize with random values
    for (int i = 0; i < 12 * N; i++) {
        clC[i] = (float)rand()/(float)(RAND_MAX);
    }

    for (int i = 0; i < 6 * N; i++) {
        tiPC[2*i] = (float)rand()/(float)(RAND_MAX);
        tiPC[2*i + 1] = 1 - tiPC[2*i];
    }

    // Copy from host to device memory
    hipMemcpy((void*) d_clC, (void*) clC, sizeof(float) * 12 * N, hipMemcpyHostToDevice);
    hipMemcpy((void*) d_tiPC, (void*) tiPC, sizeof(float) * 12 * N, hipMemcpyHostToDevice);

    // Run kernel N times (to benchmark)
    compute_parent_likelihood<<<1, T>>>(d_clP, d_clC, d_tiPC);

    // Copy from device to host memory
    hipMemcpy((void*) clP, (void*) d_clP, sizeof(float) * 6 * N, hipMemcpyDeviceToHost);

    // Print first 6
    printf("First 6 CLs: ");
    for (int i = 0; i < 6; i++) {
        printf("%f ", clP[i]);
    }
    printf("\n");

    // Validation
    int errors = 0;
    for (int t = 0; t < N; t++) {
        for (int i = 0; i < 6; i++) {
            int id = t * 6 + i;
            int l = id * 2, r = id * 2 + 1;
            if (clP[id] - (clC[l] * tiPC[l] + clC[r] * tiPC[r]) > 1e-6) {
                errors++;
            }
        }
    }
    printf("Errors: %d\n", errors);

    // Free host memory
    free(clP);
    free(clC);
    free(tiPC);

    // Free device memory
    hipFree(d_clP);
    hipFree(d_clC);
    hipFree(d_tiPC);

    return 0;
}